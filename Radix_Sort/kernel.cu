#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define BLOCK_SIZE 512 //TODO: You can change this

#define wbCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void blockadd(int* g_aux, int* g_odata, int n){
	int id = blockIdx.x*blockDim.x + threadIdx.x; //Id of the thread within the block

	if (blockIdx.x > 0 && id < n){
		g_odata[id] += g_aux[blockIdx.x];
	}

}

__global__ void split(int *in_d, int *out_d, int length, int shamt) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
	int bit = 0;

	if (index < length) {
		bit = in_d[index] & (1 << shamt);
		//bit = ~bit;

		if (bit > 0)
            bit = 1;
        else
            bit = 0;

		__syncthreads();

		out_d[index] = 1 -bit;

	}

}

__global__ void split2(int *in_d,  int *lb_d,  int length) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	int x = in_d[length - 1] + lb_d[length - 1];
	__syncthreads();

	if (index < length) {
		if (lb_d[index] == 0) {
			__syncthreads();
			int val = in_d[index];
			in_d[index] = index - val + x;

		}
	}

}

__global__ void scatter( int *in_d,  int *index_d,  int *out_d,  int length) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < length) {

		int val = index_d[index];
		__syncthreads();

		out_d[val] = in_d[index];

	}
}

__global__ void scan(int *g_odata, int *g_idata, int *g_aux, int n){

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ float temp[BLOCK_SIZE]; 

	if (i < n){
		temp[threadIdx.x] = g_idata[i];
	}

	for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2){
		__syncthreads();
		float in1 = 0.0;

		if (threadIdx.x >= stride){
			in1 = temp[threadIdx.x - stride];
		}
		__syncthreads();
		temp[threadIdx.x] += in1;
	}

	__syncthreads();

	if (i + 1 < n) g_odata[i + 1] = temp[threadIdx.x];
	g_odata[0] = 0;//MUST DO THIS TO MAKE IT WORK!!!


	if (g_aux != NULL && threadIdx.x == blockDim.x - 1){

		g_aux[blockIdx.x] = g_odata[i + 1];
		g_odata[i + 1] = 0;
	}
}
void swap(int* in, int* out){

    int *tmp;
    tmp = in;
    in = out;
    out = tmp;
}

void recursive_scan(int* deviceOutput, int* deviceInput, int numElements){
	int numBlocks = (numElements / BLOCK_SIZE) + 1;
	if (numBlocks == 1){ 
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid(numBlocks, 1);

		scan << <grid, block >> >(deviceOutput, deviceInput, NULL, numElements);
		hipDeviceSynchronize();
	}
	else{ 
		int* deviceAux;
		hipMalloc((void**)&deviceAux, (numBlocks*sizeof(int)));

		int *deviceAuxPass;
		hipMalloc((void**)&deviceAuxPass, (numBlocks*sizeof(int)));

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid(numBlocks, 1);

		scan << <grid, block >> >(deviceOutput, deviceInput, deviceAux, numElements);
		wbCheck(hipDeviceSynchronize());


		dim3 grid2(1, 1);
		dim3 block2(numBlocks, 1, 1);

		scan << <grid2, block2 >> >(deviceAuxPass, deviceAux, NULL, numBlocks);
		wbCheck(hipDeviceSynchronize());

		recursive_scan(deviceAuxPass, deviceAux, numBlocks);

		blockadd << <block2, block >> >(deviceAuxPass, deviceOutput, numElements);
		wbCheck(hipDeviceSynchronize());

		hipFree(deviceAux);
		hipFree(deviceAuxPass);
	}

}

void sort(int* deviceInput, int *deviceOutput, int numElements, int* hostInput)
{
	//TODO: Modify this to complete the functionality of the sort on the deivce
	int numBlocks = (numElements / BLOCK_SIZE) + 3;
	int *help; int *help2;


	dim3 block(BLOCK_SIZE, 1);
	dim3 grid(numBlocks, 1);

	dim3 grid2(1, 1);
	dim3 block2(numBlocks, 1, 1);

	hipMalloc(&help, sizeof(int)*numElements);
	hipMalloc(&help2, sizeof(int)*numElements);

	for (int bit = 0; bit < 15; bit++){

		split << <grid, block >> >(deviceInput, deviceOutput, numElements, bit);
		hipDeviceSynchronize();		
		
		recursive_scan(help2, deviceOutput, numElements);
		//scan << <grid, block >> >(help2, deviceOutput, NULL, numElements);
		hipDeviceSynchronize();

		/**blockadd << <block2, block >> >(help2, deviceOutput, numElements);
		hipDeviceSynchronize();**/

		split2 << <grid, block >> >(help2, deviceOutput, numElements);
		hipDeviceSynchronize();

		scatter << <grid, block >> >(deviceInput, help2, deviceOutput, numElements);
		hipDeviceSynchronize();

		//swap(deviceInput, deviceOutput);
		int *tmp;
		tmp = deviceInput;
		deviceInput = deviceOutput;
		deviceOutput= tmp;
	}

	/**int *tmp;
	tmp = deviceInput;
	deviceInput = deviceOutput;
	deviceOutput = tmp;**/
}


int main(int argc, char **argv) {
	wbArg_t args;
	int *hostInput;  // The input 1D list
	int *hostOutput; // The output list
	int *deviceInput;
	int *deviceOutput;
	int numElements; // number of elements in the list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &numElements, "integral_vector");
	hipHostAlloc(&hostOutput, numElements * sizeof(int), hipHostMallocDefault);
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ", numElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(int)));
	wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(int)));
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Clearing output memory.");
	wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(int)));
	wbTime_stop(GPU, "Clearing output memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(int),
		hipMemcpyHostToDevice));
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	wbTime_start(Compute, "Performing CUDA computation");
	sort(deviceInput, deviceOutput, numElements, hostInput);
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
		hipMemcpyDeviceToHost));
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceInput);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, numElements);

	free(hostInput);
	hipHostFree(hostOutput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
